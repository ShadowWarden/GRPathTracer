#include "hip/hip_runtime.h"
/* main.cu
 *
 *  Omkar H. Ramachandran
 *  omkar.ramachandran@colorado.edu
 *
 *  Rotating triangle : OpenGL + SDL
 */

#include "general.h"

int fps=0,sec0=0,count=0;
int width = 640;
int height = 480;
int size = width*height;
int mode = 0; // 0 = Newton, 1 = Einstein
int Mode; // RGB vs RGBA
bool is_change = true;

float *ux,*uy,*uz,*x,*y,*z;

float *dev_ux,*dev_uy,*dev_uz,*dev_x,*dev_y,*dev_z;

Uint32 * pixels;

void quit(int code){
	hipFree(dev_ux);
	hipFree(dev_uy);
	hipFree(dev_uz);
	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);

	free(ux);
	free(uy);
	free(uz);
	free(x);
	free(y);
	free(z);

	free(pixels);

	SDL_Quit();
	exit(code);
}

Uint32 get_pixel32(SDL_Surface *surface, int x, int y)
{
	int bpp = surface->format->BytesPerPixel;
	/* Here p is the address to the pixel we want to retrieve */
	Uint8 *p = (Uint8 *)surface->pixels + y * surface->pitch + x * bpp;

	switch(bpp) {
		case 1:
			return *p;
			break;

		case 2:
			return *(Uint16 *)p;
			break;

		case 3:
			if(SDL_BYTEORDER == SDL_BIG_ENDIAN)
				return p[0] << 16 | p[1] << 8 | p[2];
			else
				return p[0] | p[1] << 8 | p[2] << 16;
			break;

		case 4:
			return *(Uint32 *)p;
			break;

		default:
			return 0;       /* shouldn't happen, but avoids warnings */
	}
}

int FramesPerSecond(void){
	int sec = glutGet(GLUT_ELAPSED_TIME)/1000;
	if (sec!=sec0){
		sec0 = sec;
		fps = count;
		count=0;
	}
	count++;
	return fps;
}

__global__ void ray_solver(float * ux, float * uy, float *uz, float * x, float * y, float *z){
	// Using pretested values of h and dt
	const float h = 1.414;
	const float dt = 0.5;
	// Use multiple blocks in 1D with 1024 threads in each
	int i = blockIdx.x*1024 + threadIdx.x;

	int size = 640*480;

	/* Euler method solver - Step size is not a limiting factor
	 * here, so I'm not bothering with implementing rk4 */
	if(i<size){
		//	int flag = 0;
		float r, powr6;
		while(1){
			float r = sqrt(x[i]*x[i]+y[i]*y[i]+z[i]*z[i]);
			float powr6 = pow(r,6);


			if(r<1){
				/* If the radius is less than 1, i.e the ray falls into
				 *  the black hole, then stop */
				break;
			}else if(r>10){
				/* If ray hits skydome, then stop */
				break;
			}

			float cost = z[i]/r;
			float   k11 = dt * (-3/2.0*h*h*sqrt(1-cost*cost)*x[i]/powr6),
				k12 = dt * (-3/2.0*h*h*sqrt(1-cost*cost)*y[i]/powr6),
				k13 = dt * (-3/2.0*h*h*z[i]/powr6),	
				k14 = dt * ux[i],
				k15 = dt * uy[i],
				k16 = dt * uz[i];
				ux[i] += k11;
				uy[i] += k12;
				uz[i] += k13;
				x[i] += k14;
				y[i] += k15;
				z[i] += k16;

		}
	}
}

void draw_screen(SDL_Surface *surface, Camera C,int * fps, GLuint texture){
	//        int i;
	float time = 0.001*glutGet(GLUT_ELAPSED_TIME);
	float th = C.Getth();
	float ph = C.Getph();
	float r = C.Getr();

	float Eye_x = r*sin(th*DEGtoRAD)*cos(ph*DEGtoRAD);
	float Eye_y = r*sin(th*DEGtoRAD)*sin(ph*DEGtoRAD);
	float Eye_z = r*cos(th*DEGtoRAD);


	if(mode == 0){
		float Up_z = (th>=180.0f && th<360.0f)?-1.0f:1.0f;

		gluLookAt(Eye_x,Eye_y,Eye_z,0.0,0.0,0.0,0.0,0.0,Up_z);

		// Render black hole
		sphere(0.0,0.0,0.0,1.0);
		// Render skydome
		glEnable(GL_TEXTURE_2D);
		glBindTexture(GL_TEXTURE_2D,texture);
		sphere(1.0f,1.0f,1.0f,10.0f,false,true);
		glDisable(GL_TEXTURE_2D);
	}else if(mode == 1){
		int i,j,index;
		float ratio = width/height;
		float dphi = 90*ratio;
		float dtheta = 90;
		float delphi = dphi/width;
		float deltheta = dtheta/height;


		if(is_change){
			// Raytraced black hole image
			for(index=0;index<height;index++){
				for(j=0;j<width;j++){
					i = width*index + j;
					ux[i] = 0;
					uy[i] = 0;
					uz[i] = 1;
					x[i] =  -5.0+10.0/(width-1)*j;
					y[i] = -5.0+10.0/(height-1)*index;
					z[i] = -5.0;
				}
			}


			hipMemcpy(dev_x,x,sizeof(float)*size,hipMemcpyHostToDevice);
			hipMemcpy(dev_y,y,sizeof(float)*size,hipMemcpyHostToDevice);
			hipMemcpy(dev_z,z,sizeof(float)*size,hipMemcpyHostToDevice);
			hipMemcpy(dev_ux,ux,sizeof(float)*size,hipMemcpyHostToDevice);
			hipMemcpy(dev_uy,uy,sizeof(float)*size,hipMemcpyHostToDevice);
			hipMemcpy(dev_uz,uz,sizeof(float)*size,hipMemcpyHostToDevice);

			ray_solver<<<size/1024,1024>>>(dev_ux,dev_uy,dev_uz,dev_x,dev_y,dev_z);

			hipMemcpy(x,dev_x,sizeof(float)*size,hipMemcpyDeviceToHost);
			hipMemcpy(y,dev_y,sizeof(float)*size,hipMemcpyDeviceToHost);
			hipMemcpy(z,dev_z,sizeof(float)*size,hipMemcpyDeviceToHost);
			is_change = false;
		}
		/* We now have x, y and z. Calculate theta and phi for each
		 *  and print pixel
		 */
		for(index=0;index<height;index++){
			for(j=0;j<width;j++){
				i = width*index+j;	
				float r = sqrt(x[i]*x[i]+y[i]*y[i]+z[i]*z[i]);
				//		printf("%f\t%f\t%f\t\n",x[i],y[i],z[i]);
				if(r>10.0){
					// Ray actually hit the skydome
					float theta = acos(z[i]/r);
					if(theta>3.141592){
						theta -= 3.141592;
					}
					float phi = atan2(y[i],x[i]);
					phi += 3.14159265;
					int xcord = (phi/3.141592/2.0*surface->w);
					int ycord = (theta/3.141592*surface->h);
					pixels[i] = get_pixel32(surface,xcord,ycord);
					//	pixels[i] = 1;
					//					if(!flag){
					//						printf("%d\n",xcord);
					//					}
				}else if(r<1.0){
					pixels[i] = 0;
				}
			}
		}

		glDrawPixels(width,height,GL_RGBA,GL_UNSIGNED_BYTE,pixels);
	}
	*fps = FramesPerSecond();
	SDL_GL_SwapBuffers();
}

int main(int argc, char ** argv){
	const SDL_VideoInfo* info = NULL;

	/* Colour depth */
	int bpp = 0;
	int flags = 0;

	if(SDL_Init(SDL_INIT_VIDEO) < 0){
		fprintf(stderr, "Video init failed: %s\n",SDL_GetError());
		quit(1);
	}

	info = SDL_GetVideoInfo();

	if(!info){
		fprintf(stderr, "Video query failed: %s\n", SDL_GetError());
		quit(1);
	}

	bpp = info->vfmt->BitsPerPixel;

	flags = SDL_OPENGL;

	SDL_GL_SetAttribute( SDL_GL_RED_SIZE, 8 );
	SDL_GL_SetAttribute( SDL_GL_GREEN_SIZE, 8 );
	SDL_GL_SetAttribute( SDL_GL_BLUE_SIZE, 8 );
	SDL_GL_SetAttribute( SDL_GL_DEPTH_SIZE, 16 );
	SDL_GL_SetAttribute( SDL_GL_STENCIL_SIZE, 0 );
	SDL_GL_SetAttribute( SDL_GL_DOUBLEBUFFER, 1 );

	if(SDL_SetVideoMode(width, height, bpp, flags) == 0){
		fprintf(stderr, "Video mode set failed: %s\n", SDL_GetError());
		quit(1);
	}

	SDL_WM_SetCaption("Black Hole RayTracer","BHRT");

	ux = (float *) malloc(sizeof(float)*size);
	uy = (float *) malloc(sizeof(float)*size);
	uz = (float *) malloc(sizeof(float)*size);
	x = (float *) malloc(sizeof(float)*size);
	y = (float *) malloc(sizeof(float)*size);
	z = (float *) malloc(sizeof(float)*size);



	hipMalloc((void **) &dev_ux, sizeof(float)*size);
	hipMalloc((void **) &dev_uy, sizeof(float)*size);
	hipMalloc((void **) &dev_uz, sizeof(float)*size);
	hipMalloc((void **) &dev_x, sizeof(float)*size);
	hipMalloc((void **) &dev_y, sizeof(float)*size);
	hipMalloc((void **) &dev_z, sizeof(float)*size);



	pixels = new Uint32[height*width];
	// Load image
	//	GLuint texture;
	GLuint TextureID = 0;

	// You should probably use CSurface::OnLoad ... ;)
	//-- and make sure the Surface pointer is good!
	SDL_Surface* Surface = IMG_Load("7415.jpg");

	glGenTextures(1, &TextureID);
	glBindTexture(GL_TEXTURE_2D, TextureID);

	Mode = GL_RGB;

	//	int BPP = Surface->format->BytesPerPixel;

	if(Surface->format->BytesPerPixel == 4) {
		Mode = GL_RGBA;
	}

	printf("surface->w: %d, surface->h = %d\n",Surface->w,Surface->h);


	glTexImage2D(GL_TEXTURE_2D, 0, Mode, Surface->w, Surface->h, 0, Mode, GL_UNSIGNED_BYTE, Surface->pixels);
	glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexEnvi(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_MODULATE);

	printf("Pixel : %d\n",get_pixel32(Surface,10,10));

	Camera C;
	int fps;
	setup_opengl(width, height);

	while(1){
		render_init();
		process_events(&C,&fps);
		draw_screen(Surface,C,&fps,TextureID);
	}
	quit(0);
	return 0;
}
