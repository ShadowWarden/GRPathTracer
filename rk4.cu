/* rk4.c
*  Omkar H. Ramachandran
*
*  4th Order Runge Kutta implementation for solving an ODE.
*  ux' = -3/2*x/(x**2+y**2)**3
*  uy' = -3/2*y/(x**2+y**2)**3
*  dx/dt = ux
*  dt/dt = uy
*  f(t,ux,x,y)
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <sys/types.h>

#define Nruns 1
#define E 2.71828182845904
#define DIM 10

double calctime(struct timeval start, struct timeval end){
        double time = 0.0;
        time = end.tv_usec - start.tv_usec;
        time = time/1000000.0;
        time += end.tv_sec - start.tv_sec;

        return time;
}


__global__ void euler(float * ux, float * uy, float * x, float * y, int * n){
//	printf("%g\t%g\t%g\t%g\n",dt,t,u1,u2);
	const float h = 1.414;
	const float dt = 0.1;
	int i = blockIdx.x*1024 + threadIdx.x;
	int j=0;
	while(j!=n[0]){
		float	k11 = dt * (-3/2.0*h*h*x[i]/pow(x[i]*x[i]+y[i]*y[i],3)),
			k12 = dt * (-3/2.0*h*h*y[i]/pow(x[i]*x[i]+y[i]*y[i],3)),
			k13 = dt * ux[i],
			k14 = dt * uy[i];
			ux[i] += k11;
			uy[i] += k12;
			x[i] += k13;
			y[i] += k14;
		if(sqrt(x[i]*x[i]+y[i]*y[i])<1){
	//		x[i] -= k13;
	//		y[i] -= k14;
			break;
		}else if(sqrt(x[i]*x[i]+y[i]*y[i])>10){
	//		x[i] -= k13;
	//		y[i] -= k14;
			break;
		}
		j++;
	}
//	sol[1] = u2 + (k12+2*k22+2*k32+k42)/6;	
//	printf("%g\t%g\n",sol[0],sol[1]);
}

int main(int argc, char ** argv){
	double x0 = 0, x1 = 10, dx = 0.1;
	int n = 1 + (x1 - x0)/dx;
	int i;
	int N = 1024;
	float *ux = (float *) malloc(sizeof(float)*N);
	float *uy = (float *) malloc(sizeof(float)*N);
	float *x = (float *) malloc(sizeof(float)*N);
	float *y = (float *) malloc(sizeof(float)*N); 

        double time;
        struct timeval start;
        struct timeval end;

	for(i=0;i<N;i++){	
		ux[i] = 0;
		uy[i] = 1;	
		x[i] = -4.0/N*i;
		y[i] = -5.0;
	}

	float * dev_ux, * dev_uy, * dev_x, * dev_y;
	int * dev_n;

	hipMalloc((void **) &dev_ux, sizeof(float)*N);
	hipMalloc((void **) &dev_uy, sizeof(float)*N);
	hipMalloc((void **) &dev_x, sizeof(float)*N);
	hipMalloc((void **) &dev_y, sizeof(float)*N);
	hipMalloc((void **) &dev_n, sizeof(int));

	hipMemcpy(dev_ux,ux,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(dev_uy,uy,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(dev_x,x,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(dev_y,y,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(dev_n,&n,sizeof(int),hipMemcpyHostToDevice);

	gettimeofday(&start, NULL);
	euler<<<1,N>>>(dev_ux,dev_uy,dev_x,dev_y,dev_n);
	gettimeofday(&end, NULL);

	hipMemcpy(ux,dev_ux,sizeof(float)*N,hipMemcpyDeviceToHost);
	hipMemcpy(uy,dev_uy,sizeof(float)*N,hipMemcpyDeviceToHost);
	hipMemcpy(x,dev_x,sizeof(float)*N,hipMemcpyDeviceToHost);
	hipMemcpy(y,dev_y,sizeof(float)*N,hipMemcpyDeviceToHost);
//	cudaMemcpy(&n,dev_n,sizeof(int),cudaMemcpyDeviceToHost);


	hipFree(dev_ux);
	hipFree(dev_uy);
	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_n);
	
	time = calctime(start,end);
	
	printf("(Nruns = %d) Calctime: %lf\n",Nruns,time);	
	for(i=0;i<N;i++)
		printf("%g\t%g\t%g\t%g\n", ux[i], uy[i], x[i], y[i]);
 
	free(x);
	free(y);
	free(ux);
	free(uy);
	return 0;
}
